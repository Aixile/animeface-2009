#include "hip/hip_runtime.h"
#include "nv_face_detect.h"
#include "nv_face_feature.h"
#include "nv_face_detect_internal.h"
#include <cutil.h>
#include <hip/hip_runtime.h>

#define NV_INTEGRAL_V(sum, x, y, xw, yh) \
(NV_MAT3D_V((sum), (yh), (xw), 0) \
- NV_MAT3D_V((sum), (yh), (x), 0) \
- (NV_MAT3D_V((sum), (y), (xw), 0) - NV_MAT3D_V((sum), (y), (x), 0))) 

#define NV_FACE_FEATURE_MAX 30000

__global__
void
feature_kernel(nv_matrix_t *feature, 
			   nv_matrix_t *gray_integral,
			   const nv_rect_t *rect, int data_step, int data_m);
__global__
void
dir_kernel(nv_matrix_t *feature, int data_m,
		   const nv_matrix_t *input_w, 
		   const nv_matrix_t *hidden_w, 
		   const nv_matrix_t *input_bias, 
		   const nv_matrix_t *hidden_bias, 
		   nv_matrix_t *input_y, 
		   nv_matrix_t *hidden_y,
		   nv_matrix_t *flags
		   );
__global__
void
face_kernel(nv_matrix_t *feature, 
		   const int *idx, int nidx,
		   const nv_matrix_t *input_w, 
		   const nv_matrix_t *hidden_w, 
		   const nv_matrix_t *input_bias, 
		   const nv_matrix_t *hidden_bias, 
		   nv_matrix_t *input_y, 
		   nv_matrix_t *hidden_y,
		   nv_matrix_t *flags
		   );


__host__
static void nv_alloc_device_windows(nv_rect_t **rects, int *nrect,
									const nv_matrix_t *edge_integral,
									float base, float start_scale, float up_scale, 
									int width, int height)
{
	float scale = start_scale;
	float xs = 0.0f;
	float ys = 0.0f;
	int count = 0;
	nv_rect_t *candidates;

	while (NV_MIN(width, height) / scale > base) {
		int yi, ye;
		int window = (int)(32.0f * scale);
		ye = (height - (base * scale) - ys) / (4.0f * scale);
		for (yi = 0; yi < ye; ++yi) {
			int y = (int)(ys + (yi * 4.0f * scale));
			int xi, xe;
			xe = (width - (32.0f * scale) - xs) / (4.0f * scale);
			for (xi = 0; xi < xe; ++xi) {
				++count;
			}
		}
		scale *= up_scale;
	}
	candidates = (nv_rect_t *)malloc(sizeof(nv_rect_t) * count);
	count = 0;
	scale = start_scale;

	while (NV_MIN(width, height) / scale > base) {
		int yi, ye;
		int window = (int)(32.0f * scale);
		ye = (height - (base * scale) - ys) / (4.0f * scale);
		for (yi = 0; yi < ye; ++yi) {
			int y = (int)(ys + (yi * 4.0f * scale));
			int xi, xe;
			xe = (width - (32.0f * scale) - xs) / (4.0f * scale);
			for (xi = 0; xi < xe; ++xi) {
				int x = (int)(xs + (xi * 4.0f * scale));
				int px = x;
				int py = y;
				int ex = (x + ((scale * 32.0f) + 0.5f));
				int ey = (y + ((scale * 32.0f) + 0.5f));
				float area = NV_MAT3D_V(edge_integral, ey, ex, 0)
					- NV_MAT3D_V(edge_integral, ey, px, 0)
					- (NV_MAT3D_V(edge_integral, py, ex, 0) - NV_MAT3D_V(edge_integral, py, px, 0));
				if (!nv_is_face_edge(window, scale, area)) {
					continue;
				}
				candidates[count].x = px;
				candidates[count].y = py;
				candidates[count].width = ex - px;
				candidates[count].height = ey - py;
				++count;
			}
		}
		scale *= up_scale;
	}

	// cuda malloc
	*rects = candidates;
	*nrect = count;
}

int nv_face_detect_gpu(const nv_mlp_t **mlp, int nmlp,
							const nv_mlp_t *dir_mlp, const nv_mlp_t *parts_mlp,
							const nv_matrix_t *gray_integral, 
							const nv_matrix_t *edge_integral, 
							const nv_rect_t *image_size,
							nv_face_position_t *face_pos, 
							int maxface)
{
	nv_rect_t *rects, *dev_rects;
	int nrect, i, flag_m;
	int data_m, data_m_all, data_step;
	nv_matrix_t *dev_feature;
	nv_matrix_t *dev_integral;
	nv_matrix_t *flags;
	nv_matrix_t *label;
	int threads, blocks;
	nv_mlp_t kdir_mlp, kface_mlp;
	nv_matrix_t *kdir_iy, *kdir_hy, *kflags;
	nv_matrix_t *kface_iy, *kface_hy;
	nv_matrix_t *feature = nv_matrix_alloc(NV_FACE_HAARLIKE_DIM, NV_FACE_FEATURE_MAX);
	float *kflags_v;
	int *kidx, idx[NV_FACE_FEATURE_MAX], nidx;
	int face_count;
	int t=nv_clock();
	nv_matrix_t *mat_t;

	CUDA_SAFE_CALL(hipMalloc((void **)&kidx, sizeof(int) * NV_FACE_FEATURE_MAX));
	dev_feature = nv_gpu_matrix_alloc(NULL, NV_FACE_FEATURE_MAX, NV_FACE_HAARLIKE_DIM);
	dev_integral = nv_gpu_matrix_copy(gray_integral);

	kdir_mlp.input_w = nv_gpu_matrix_copy(dir_mlp->input_w);
	kdir_mlp.input_bias = nv_gpu_matrix_copy(dir_mlp->input_bias);
	kdir_mlp.hidden_w = nv_gpu_matrix_copy(dir_mlp->hidden_w);
	kdir_mlp.hidden_bias = nv_gpu_matrix_copy(dir_mlp->hidden_bias);
	kdir_iy = nv_gpu_matrix_alloc(NULL, NV_FACE_FEATURE_MAX, dir_mlp->input_w->m);
	kdir_hy = nv_gpu_matrix_alloc(NULL, NV_FACE_FEATURE_MAX, dir_mlp->hidden_w->m);
	kface_mlp.input_w = nv_gpu_matrix_copy(mlp[0]->input_w);
	kface_mlp.input_bias = nv_gpu_matrix_copy(mlp[0]->input_bias);
	kface_mlp.hidden_w = nv_gpu_matrix_copy(mlp[0]->hidden_w);
	kface_mlp.hidden_bias = nv_gpu_matrix_copy(mlp[0]->hidden_bias);
	kface_iy = nv_gpu_matrix_alloc(NULL, NV_FACE_FEATURE_MAX, mlp[0]->input_w->m);
	kface_hy = nv_gpu_matrix_alloc(NULL, NV_FACE_FEATURE_MAX, mlp[0]->hidden_w->m);
	kflags = nv_gpu_matrix_alloc(&kflags_v, 1, NV_FACE_FEATURE_MAX);

	nv_alloc_device_windows(&rects, &nrect,
		edge_integral,
		32.0f, 1.44f, 1.095f,
		image_size->width, image_size->height);

	CUDA_SAFE_CALL(hipMalloc((void **)&dev_rects, sizeof(nv_rect_t) * nrect));
	CUDA_SAFE_CALL(hipMemcpy(dev_rects, rects, sizeof(nv_rect_t) * nrect, hipMemcpyHostToDevice));

	flags = nv_matrix_alloc(1, nrect);
	nv_matrix_zero(flags);

	data_step = 0;
	flag_m = 0;
	data_m_all = nrect;
	while (data_m_all > 0) {
		if (data_m_all > NV_FACE_FEATURE_MAX) {
			data_m = NV_FACE_FEATURE_MAX;
		} else {
			data_m = data_m_all;
		}
		// face feature kernel
		blocks = nv_gpu_block(data_m);
		threads = nv_gpu_thread(data_m);
		t = nv_clock();

		// �������o
		feature_kernel<<<blocks, threads>>>(
			dev_feature, 
			dev_integral, 
			dev_rects, NV_FACE_FEATURE_MAX * data_step, data_m
		);
		CUT_CHECK_ERROR("feature_kernel() execution failed\n");
		hipDeviceSynchronize();
		//printf("feature extraction kernel: %d\n", nv_clock() -t);
		//t = nv_clock();

		// ��������� 0 = flag 1
		dir_kernel<<<blocks, threads>>>(
			dev_feature, data_m,
			kdir_mlp.input_w,
			kdir_mlp.hidden_w,
			kdir_mlp.input_bias,
			kdir_mlp.hidden_bias,
			kdir_iy, kdir_hy,
			kflags
		);
		CUT_CHECK_ERROR("dir_kernel() execution failed\n");
		hipDeviceSynchronize();
		//printf("direction kernel: %d\n", nv_clock() -t);
		//t = nv_clock();
		CUDA_SAFE_CALL(hipMemcpy(
			&NV_MAT_V(flags, NV_FACE_FEATURE_MAX * data_step, 0), 
			kflags_v, sizeof(float) * data_m, 
			hipMemcpyDeviceToHost));
		//printf("flag tran: %d\n", nv_clock() -t);
		//t = nv_clock();

		// �画��
		nidx = 0;
		for (i = 0; i < data_m; ++i) {
			if (NV_MAT_V(flags, NV_FACE_FEATURE_MAX * data_step + i, 0) == 1.0f) {
				idx[nidx] = i;
				++nidx;
			}
		}
		//printf("flag: %d\n", nv_clock() -t);
		//t = nv_clock();
		CUDA_SAFE_CALL(hipMemcpy(kidx, idx, sizeof(int) * nidx, hipMemcpyHostToDevice));
		blocks = nv_gpu_block(nidx);
		threads = nv_gpu_thread(nidx);
		face_kernel<<<blocks, threads>>>(
			dev_feature,
			kidx, nidx,
			kface_mlp.input_w,
			kface_mlp.hidden_w,
			kface_mlp.input_bias,
			kface_mlp.hidden_bias,
			kface_iy, kface_hy,
			kflags
			);
		CUT_CHECK_ERROR("face_kernel() execution failed\n");
		hipDeviceSynchronize();
		//printf("face kernel: %d\n", nv_clock() -t);
		//t = nv_clock();
		CUDA_SAFE_CALL(hipMemcpy(
			&NV_MAT_V(flags, NV_FACE_FEATURE_MAX * data_step, 0), 
			kflags_v, sizeof(float) * data_m, 
			hipMemcpyDeviceToHost));
		//printf("flag tran: %d\n", nv_clock() -t);
		data_m_all -= data_m;
		++data_step;
	}
	face_count = 0;
	for (i = 0; i < nrect; ++i) {
		if (NV_MAT_V(flags, i, 0) == 1.0f) {
			++face_count;
		}
	}
	//printf("face: %d (%d)\n", face_count, nv_clock()-t);

	return 0;
}

__device__
static float nv_face_feature_filter2_gpu(int type,
									const nv_matrix_t *sum,
									int px, int py,
									float xscale, float yscale)
{
	int ix, iy;
	int i = 0;
	float p = 0.0f, p1 = 0.0f, p2 = 0.0f;
	int area_all = 0;
	int ystep = (int)(1.0f * yscale + 0.5f);
	int xstep = (int)(1.0f * xscale + 0.5f);

	if (type == 1) {
		// |�_|
		for (i = 0; i < 7; ++i) {
			int ppx = px + (int)((1.0f + i) * xscale + 0.5f);
			int ppy = py + (int)(i * yscale + 0.5f);
			int eex = px + (int)(8.0f * xscale + 0.5f);
			int eey = py + (int)((i + 1) * yscale + 0.5f);

			//printf("p1: %d, %d, %d, %d\n", 1+i,8,i,i+1);

			p1 += NV_INTEGRAL_V(sum, ppx, ppy, eex, eey);
			area_all += (eex - ppx) * (eey - ppy);	
		}
		for (i = 1; i < 8; ++i) {
			int ppx = px + (int)(0.0f * xscale + 0.5f);
			int ppy = py + (int)(i * yscale + 0.5f);
			int eex = px + (int)(i * xscale + 0.5f);
			int eey = py + (int)((i + 1) * yscale + 0.5f);

			//printf("p2: %d, %d, %d, %d\n", 0,i,i,i+1);
			p2 += NV_INTEGRAL_V(sum, ppx, ppy, eex, eey);
			area_all += (eex - ppx) * (eey - ppy);	
		}
		p = (p1 - p2) / (area_all * 255.0f);
	} else {
		// |/|
		for (i = 0; i < 7; ++i) {
			int ppx = px + (int)(0.0f * xscale + 0.5f);
			int ppy = py + (int)(i * yscale + 0.5f);
			int eex = px + (int)((7.0f - i) * xscale + 0.5f);
			int eey = py + (int)((i + 1) * yscale + 0.5f);

			//printf("p1: %d, %d, %d, %d\n", 0, 7-i, i, i+1);
			
			p1 += NV_INTEGRAL_V(sum, ppx, ppy, eex, eey);
			area_all += (eex - ppx) * (eey - ppy);	
		}
		for (i = 1; i < 8; ++i) {
			int ppx = px + (int)((8.0f - i) * xscale + 0.5f);
			int ppy = py + (int)(i * yscale + 0.5f);
			int eex = px + (int)(8.0f * xscale + 0.5f);
			int eey = py + (int)((i + 1) * yscale + 0.5f);
			//printf("p2: %d, %d, %d, %d\n", 8-i, 8, i, i+1);

			p2 += NV_INTEGRAL_V(sum, ppx, ppy, eex, eey);
			area_all += (eex - ppx) * (eey - ppy);	
		}
		p = (p1 - p2) / (area_all * 255.0f);
	}

	return p;
}


__device__
float nv_face_feature_filter_gpu(const float *filter,
								 const nv_matrix_t *sum,
								 int px, int py,
								 float xscale, float yscale)
{
	int ix, iy;
	int i = 0;
	float p = 0.0f;
	int area_all = 0;
	int ystep = (int)(1.0f * yscale + 0.5f);
	int xstep = (int)(1.0f * xscale + 0.5f);
	return 0.0f;

	for (iy = 0; iy < 8; ++iy) {
		int ppy = py + (int)(iy * yscale + 0.5f);
		int eey = ppy + ystep;
		for (ix = 0; ix < 8; ++ix) {
			int ppx = px + (int)(ix * xscale + 0.5f);
			int eex = ppx + xstep;
			float area = NV_MAT3D_V(sum, eey, eex, 0)
				- NV_MAT3D_V(sum, eey, ppx, 0)
				- (NV_MAT3D_V(sum, ppy, eex, 0) - NV_MAT3D_V(sum, ppy, ppx, 0));

			p += area * filter[i];
			if (filter[i] != 0.0f) {
				area_all += (eey - ppy) * (eex - ppx);
			}
			++i;
		}
	}
	p /= area_all * 255.0f;
	return p;
}

__constant__
const float filter_diagonal1_gpu[] = {
	0.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
	-1.0f, 0.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
	-1.0f, -1.0f, 0.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f,
	-1.0f, -1.0f, -1.0f, 0.0f, 1.0f, 1.0f, 1.0f, 1.0f,
	-1.0f, -1.0f, -1.0f, -1.0f, 0.0f, 1.0f, 1.0f, 1.0f,
	-1.0f, -1.0f, -1.0f, -1.0f, -1.0f, 0.0f, 1.0f, 1.0f,
	-1.0f, -1.0f, -1.0f, -1.0f, -1.0f, -1.0f, 0.0f, 1.0f,
	-1.0f, -1.0f, -1.0f, -1.0f, -1.0f, -1.0f, -1.0f, 0.0f,
};
__constant__
const float filter_diagonal2_gpu[] = {
	1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 0.0f,
	1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 0.0f, -1.0f,
	1.0f, 1.0f, 1.0f, 1.0f, 1.0f, 0.0f, -1.0f, -1.0f,
	1.0f, 1.0f, 1.0f, 1.0f, 0.0f, -1.0f, -1.0f, -1.0f,
	1.0f, 1.0f, 1.0f, 0.0f, -1.0f, -1.0f, -1.0f, -1.0f,
	1.0f, 1.0f, 0.0f, -1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
	1.0f, 0.0f, -1.0f, -1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
	0.0f, -1.0f, -1.0f, -1.0f, -1.0f, -1.0f, -1.0f, -1.0f,
};

__device__
void nv_face_feature_gpu(nv_face_haarlike_normalize_e normalize_type,
						 nv_matrix_t *feature, 
						 int feature_m,
						 const nv_matrix_t *sum,
						 int x, int y, int width, int height,
						 __shared__ float *shd_mem, int shd_idx)
{
	int ix, iy, n;
	int py, px, ey, ex, sy, sx, hy, hx;
	float p1, p2, area;
	float scale_area;
	float ey_ex, ey_px, py_ex, py_px;
	float v, vmax, vmin;
	float xscale = width / 32.0f;
	float yscale = height / 32.0f;
	float ystep = yscale;
	float xstep = xscale;
	int hystep = (32 - 8) / 2 * 8;

	// level1
	for (iy = 0; iy < 32-8; iy += 2) {
		py = y + (int)(ystep * iy + 0.5f);
		ey = py + (int)(8.0f * ystep + 0.5f);
		sy = (int)(4.0f * ystep + 0.5f);
		hy = iy / 2;
		for (ix = 0; ix < 32-8; ix += 2) {
			px = x + (int)(xstep * ix + 0.5f);
			ex = px + (int)(8.0f * xstep + 0.5f);
			sx = (int)(4.0f * xstep + 0.5f);
			hx = ix / 2;
			scale_area = 1.0f / ((ex - px) * (ey - py) * 255.0f);
			ey_ex = NV_MAT3D_V(sum, ey, ex, 0);
			ey_px = NV_MAT3D_V(sum, ey, px, 0);
			py_ex = NV_MAT3D_V(sum, py, ex, 0);
			py_px = NV_MAT3D_V(sum, py, px, 0);

			// �S�G���A
			area = ey_ex - ey_px - (py_ex - py_px);

			// 1
			// [ ]
			// [ ]
			p1 = NV_MAT3D_V(sum, py + sy, ex, 0)
				   - NV_MAT3D_V(sum, py + sy, px, 0)
				   - (py_ex - py_px);
			p2 = area - p1;
			if (p1 > p2) {
				shd_mem[NV_GPU_THREAD_MAX * 0 + shd_idx] = (p1 - p2) * scale_area;
				shd_mem[NV_GPU_THREAD_MAX * 1 + shd_idx] = 0.0f;
			} else {
				shd_mem[NV_GPU_THREAD_MAX * 0 + shd_idx] = 0.0f;
				shd_mem[NV_GPU_THREAD_MAX * 1 + shd_idx] = (p2 - p1) * scale_area;
			}

			// 2
			// [ ][ ]
			p1 = NV_MAT3D_V(sum, ey, px + sx, 0)
				- ey_px
				- (NV_MAT3D_V(sum, py, px + sx, 0) - py_px);
			p2 = area - p1;
			if (p1 > p2) {
				shd_mem[NV_GPU_THREAD_MAX * 2 + shd_idx] = (p1 - p2) * scale_area;
				shd_mem[NV_GPU_THREAD_MAX * 3 + shd_idx] = 0.0f;
			} else {
				shd_mem[NV_GPU_THREAD_MAX * 2 + shd_idx] = 0.0f;
				shd_mem[NV_GPU_THREAD_MAX * 3 + shd_idx] = (p2 - p1) * scale_area;
			}

			// 3
			// |/|
			p1 = nv_face_feature_filter2_gpu(1, sum, px, py, xscale, yscale);
			if (p1 > 0.0f) {
				shd_mem[NV_GPU_THREAD_MAX * 4 + shd_idx] = p1;
				shd_mem[NV_GPU_THREAD_MAX * 5 + shd_idx] = 0.0f;
			} else {
				shd_mem[NV_GPU_THREAD_MAX * 4 + shd_idx] = 0.0f;
				shd_mem[NV_GPU_THREAD_MAX * 5 + shd_idx] = -1.0f * p1;
			}

			// 4
			// |�_|
			p1 = nv_face_feature_filter2_gpu(2, sum, px, py, xscale, yscale);
			if (p1 > 0.0f) {
				shd_mem[NV_GPU_THREAD_MAX * 6 + shd_idx] = p1;
				shd_mem[NV_GPU_THREAD_MAX * 7 + shd_idx] = 0.0f;
			} else {
				shd_mem[NV_GPU_THREAD_MAX * 6 + shd_idx] = 0.0f;
				shd_mem[NV_GPU_THREAD_MAX * 7 + shd_idx] = -1.0f * p1;
			}
			
			for (n = 0; n < 8; ++n) {
				NV_MAT_VT(feature, feature_m, hy * hystep + hx * 8 + n) = shd_mem[NV_GPU_THREAD_MAX * n + shd_idx];
			}
		}
	}
	
	// ���K��
	switch (normalize_type) {
	case NV_NORMALIZE_MAX:
		// �ő�l=1.0
		vmax = 0.0f;
		vmin = FLT_MAX;
		for (n = 0; n < feature->m; ++n) {
			if (NV_MAT_VT(feature, feature_m, n) > vmax) {
				vmax = NV_MAT_VT(feature, feature_m, n);
			}
			if (NV_MAT_VT(feature, feature_m, n) != 0.0f
				&& NV_MAT_VT(feature, feature_m, n) < vmin) 
			{
				vmin = NV_MAT_VT(feature, feature_m, n);
			}
		}
		if (vmax != 0.0f && vmax > vmin) {
			v = 1.0f / (vmax - vmin);
			for (n = 0; n < feature->m; ++n) {
				if (NV_MAT_VT(feature, feature_m, n) != 0.0f) {
					NV_MAT_VT(feature, feature_m, n) = (NV_MAT_VT(feature, feature_m, n) - vmin) * v;
				}
			}
		}
		break;
	case NV_NORMALIZE_NORM:
		// �x�N�g���m����=1.0
		v = 0.0f;
		for (n = 0; n < feature->m; ++n) {
			v += NV_MAT_VT(feature, feature_m, n) * NV_MAT_VT(feature, feature_m, n);
		}
		if (v != 0.0) {
			v = 1.0f / sqrtf(v);
			for (n = 0; n < feature->m; ++n) {
				NV_MAT_VT(feature, feature_m, n) *= v;
			}
		}
		break;
	case NV_NORMALIZE_NONE:
	default:
		break;
	}
}

__device__
void nv_mlp_predict_gpu(const nv_matrix_t *input_w, 
						 const nv_matrix_t *hidden_w, 
						 const nv_matrix_t *input_bias, 
						 const nv_matrix_t *hidden_bias, 
						 int y_idx,
						 nv_matrix_t *input_y, 
						 nv_matrix_t *hidden_y, 
						 const nv_matrix_t *x, int xm,
						 float *shd_mem, int shd_idx)
{
	int n, m;

	// ���͑w
	for (m = 0; m < input_w->m; ++m) {
		float y = NV_MAT_V(input_bias, m, 0);
		for (n = 0; n < input_w->n; ++n) {
			y += NV_MAT_VT(x, xm, n) * NV_MAT_V(input_w, m, n);
		}
		NV_MAT_VT(input_y, y_idx, m) = 1.0f / (1.0f + expf(-y));
	}

	// ���ԑw
	for (m = 0; m < hidden_w->m; ++m) {
		float y = NV_MAT_V(hidden_bias, m, 0);
		for (n = 0; n < hidden_w->n; ++n) {
			y += NV_MAT_VT(input_y, y_idx, n) * NV_MAT_V(hidden_w, m, n);
		}
		NV_MAT_VT(hidden_y, y_idx, m) = 1.0f / (1.0f + expf(-y));
	}
}

__global__
void
feature_kernel(nv_matrix_t *feature, 
			   nv_matrix_t *gray_integral,
			   const nv_rect_t *rect, int data_step, int data_m)
{
	__shared__ float shd_mem[NV_GPU_THREAD_MAX * 8];
	int shd_idx = threadIdx.x;
	int my_m = blockDim.x * blockIdx.x + threadIdx.x;
	if (my_m < data_m) {
		__syncthreads();

		nv_face_feature_gpu(
			NV_NORMALIZE_MAX, 
			feature, my_m, 
			gray_integral,
			rect[data_step + my_m].x, rect[data_step + my_m].y,
			rect[data_step + my_m].width, rect[data_step + my_m].height,
			shd_mem, shd_idx);
	}
}

__global__
void
dir_kernel(nv_matrix_t *feature, int data_m,
		   const nv_matrix_t *input_w, 
		   const nv_matrix_t *hidden_w, 
		   const nv_matrix_t *input_bias, 
		   const nv_matrix_t *hidden_bias, 
		   nv_matrix_t *input_y, 
		   nv_matrix_t *hidden_y,
		   nv_matrix_t *flags
		   )
{
	__shared__ float shd_mem[NV_FACE_HAARLIKE_DIM * 2];
	int my_m = blockDim.x * blockIdx.x + threadIdx.x;
	int shd_idx = threadIdx.x;
	float mp;
	int l, n;

	if (my_m < data_m) {
		__syncthreads();
		nv_mlp_predict_gpu(
			input_w, hidden_w, 
			input_bias, hidden_bias,
			my_m,
			input_y, hidden_y,
			feature, my_m,
			shd_mem, shd_idx);
		l = -1; // nega

		for (n = 0; n < hidden_y->m; ++n) {
			if (NV_MAT_VT(hidden_y, my_m, n) > 0.5f
				&&
				mp < NV_MAT_VT(hidden_y, my_m, n)) 
			{
				mp = NV_MAT_VT(hidden_y, my_m, n);
				l = n;
			}
		}

		if (l == 0) {
			NV_MAT_V(flags, my_m, 0) = 1.0f;
		} else {
			NV_MAT_V(flags, my_m, 0) = 0.0f;
		}
	}
}

__global__
void
face_kernel(nv_matrix_t *feature, 
		   const int *idx, int nidx,
		   const nv_matrix_t *input_w, 
		   const nv_matrix_t *hidden_w, 
		   const nv_matrix_t *input_bias, 
		   const nv_matrix_t *hidden_bias, 
		   nv_matrix_t *input_y, 
		   nv_matrix_t *hidden_y,
		   nv_matrix_t *flags
		   )
{
	__shared__ float shd_mem[NV_FACE_HAARLIKE_DIM * 2];
	int shd_idx = threadIdx.x;
	int my_n = blockDim.x * blockIdx.x + threadIdx.x;
	if (my_n < nidx) {
		int my_m = idx[my_n];

		__syncthreads();
		nv_mlp_predict_gpu(
			input_w, hidden_w, 
			input_bias, hidden_bias,
			my_n,
			input_y, hidden_y,
			feature, my_m,
			shd_mem, shd_idx);

		if (NV_MAT_VT(hidden_y, my_n, 0) > 0.01f) {
			NV_MAT_V(flags, my_m, 0) = 1.0f;
		} else {
			NV_MAT_V(flags, my_m, 0) = 0.0f;
		}
	}
}

