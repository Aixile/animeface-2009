#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <cutil.h>
#include "nv_core.h"
#include "nv_ip_gray_gpu.h"

static __global__ 
void nv_gray_kernel(float *gray, const float *bgr, int gray_m);


void nv_gray_gpu(nv_matrix_t *gray, const nv_matrix_t *bgr)
{
	int blocks = nv_gpu_block(gray->m);
	int threads = nv_gpu_thread(gray->m);
	int gray_size = sizeof(float) * gray->m * gray->step;
	int bgr_size = sizeof(float) * bgr->m * bgr->step;
	float *dev_gray;
	float *dev_bgr;

	hipMalloc((void **)&dev_gray, gray_size);
	hipMalloc((void **)&dev_bgr, bgr_size);
	hipMemcpy(dev_gray, gray->v, gray_size, hipMemcpyHostToDevice);
	hipMemcpy(dev_bgr, bgr->v, bgr_size, hipMemcpyHostToDevice);
	nv_gray_kernel<<<blocks, threads, sizeof(float) * 3 * NV_GPU_THREAD_MAX>>>(
		dev_gray, dev_bgr, gray->m
	);
	hipMemcpy(gray->v, dev_gray, gray_size, hipMemcpyDeviceToHost);
	CUT_CHECK_ERROR("nv_gray_kernel() execution failed\n");

	hipFree(dev_gray);
	hipFree(dev_bgr);
}


__global__ 
void nv_gray_kernel(float *gray, const float *bgr, int gray_m)
{
	extern __shared__ float shared_mem[];
	const int my_m = blockDim.x * blockIdx.x + threadIdx.x;
	const int my_m3 = my_m * 3;
	const int shd_idx = threadIdx.x;
	float g;

	if (my_m >= gray_m) {
		return;
	}
	__syncthreads();
	shared_mem[shd_idx + 0] = bgr[my_m3 + 0];
	shared_mem[shd_idx + NV_GPU_THREAD_MAX] = bgr[my_m3 + 1];
	shared_mem[shd_idx + NV_GPU_THREAD_MAX * 2] = bgr[my_m3 + 2];
	g = NV_GRAY_B_RATE * shared_mem[shd_idx + 0];
	g += NV_GRAY_G_RATE * shared_mem[shd_idx + NV_GPU_THREAD_MAX];
	g += NV_GRAY_R_RATE * shared_mem[shd_idx + NV_GPU_THREAD_MAX * 2];
	gray[my_m] = g;
}
